#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Executed on Device, Call on Device:
__device__ int mandel(float C1, float C2, int count){
	float Z1 = C1, Z2 = C2;
	// TODO
	// Extended ANSI C
	// Each iteration as one thread.
	int i;
	for(i = 0; i < count; i++){
		if((Z1*Z1 + Z2*Z2) > 4.f)break;
		float local1 = Z1*Z1 - Z2*Z2;
		float local2 = 2.f*Z1 * Z2;
		Z1 = C1 + local1;
		Z2 = C2 + local2;
	}
	return i;
}
// Executed on Device, Call on Host 
__global__ void mandelKernel(int *devices, float X0, float Y0, float dX, float dY, int rest_X, int rest_Y, int MaxIteration, int Group) {
    // TODO : New thing is To organize the group size cut
    // To avoid error caused by the floating number, use the following pseudo code
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i, j;
    // threadIdx : the index of threads
    // blockIdx : the index of block of threads
    // blockDim : the size of each thread blocks
    // dx = (x1 - x0)/width; dy = (y1 - y0)/length;
    // here compute i and j
    i = (threadIdx.x + blockIdx.x * blockDim.x)*Group;
    j = (threadIdx.y + blockIdx.y * blockDim.y)*Group;
    // dx = (x1 - x0)/width; dy = (y1 - y0)/length;
    // X = x + row*dx, Y = y + col*dy;
    int I, J;
    float tempx, tempy;
    for(I = i; I < i + Group; I++){
	    for(J = j; J < j + Group; J++){
                if(I >= rest_X || J >= rest_Y){continue;}
                tempx = X0 + I*dX;
                tempy = Y0 + J*dY;
    // mandel can be called by device. (the actual execute result)
                int result = mandel(tempx, tempy, MaxIteration);
                devices[rest_X*J + I] = result;
		}
    }
}

//// Host code
// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
 
    // TODO : Use cudaMallocPitch and cudaHostAlloc and try different size
    int *DEVICE;
    dim3 TPB(16, 16);
    int G = 4;
    // resX = 1600, rexY = 1200
    dim3 NB(ceil(resX*1.0/(TPB.x*G)), ceil(resY*1.0/(TPB.y*G)));
    int *HOST;
    // Define for cudaMallocPitch and cudaHostAlloc
    size_t pitch;

    // allocate cude memory
    //cudaMalloc(&DEVICE, resX * resY * sizeof(int));
    hipHostAlloc(&HOST, sizeof(int)*resX*resY, hipHostMallocMapped);
    hipMallocPitch(&DEVICE, &pitch, resX*sizeof(int), resY);
    // call the global function, which is computing the result
    mandelKernel <<< NB, TPB >>> (DEVICE, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations, G);
    // the cuda's synchronization
    hipDeviceSynchronize();
    // move the result from GPU to CPU
    hipMemcpy(HOST, DEVICE, resX*resY*sizeof(int), hipMemcpyDeviceToHost);
    
    // Put the result in the output
    hipMemcpy(img, HOST, resX*resY*sizeof(int), hipMemcpyHostToHost);

    hipFree(DEVICE);
    hipHostFree(HOST);
}
